#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>

#include <time.h>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>

#define X_SIZE 10240
#define Y_SIZE 16384

#define ARRAY_SIZE (X_SIZE*Y_SIZE)

#define BLOCK_SIZE_X 32
#define BLOCK_SIZE_Y 32

#define TIMESTEPS 1000

const char* input_file_name = "input.dat";
const char* output_file_name = "output.dat";

void prtdat(int nx, int ny, float *current, const char *fnam);
void inidat(int nx, int ny, float *u);

void printDevProp(hipDeviceProp_t devProp)
{
	printf("Major revision number:         %d\n", devProp.major);
	printf("Minor revision number:         %d\n", devProp.minor);
	printf("Name:                          %s\n", devProp.name);
	printf("Total global memory:           %u or %uKB or %uMB\n", devProp.totalGlobalMem, devProp.totalGlobalMem/1024, devProp.totalGlobalMem / (1024*1024), devProp.totalGlobalMem / 1024 / 1024 / 1024);
	printf("Total shared memory per block: %u\n", devProp.sharedMemPerBlock);
	printf("Total registers per block:     %d\n", devProp.regsPerBlock);
	printf("Warp size:                     %d\n", devProp.warpSize);
	printf("Maximum memory pitch:          %u\n", devProp.memPitch);
	printf("Maximum threads per block:     %d\n", devProp.maxThreadsPerBlock);
	for (int i = 0; i < 3; ++i)
		printf("Maximum dimension %d of block:  %d\n", i, devProp.maxThreadsDim[i]);
	for (int i = 0; i < 3; ++i)
		printf("Maximum dimension %d of grid:   %d\n", i, devProp.maxGridSize[i]);
	printf("Clock rate:                    %d\n", devProp.clockRate);
	printf("Total constant memory:         %u\n", devProp.totalConstMem);
	printf("Texture alignment:             %u\n", devProp.textureAlignment);
	printf("Concurrent copy and execution: %s\n", (devProp.deviceOverlap ? "Yes" : "No"));
	printf("Number of multiprocessors:     %d\n", devProp.multiProcessorCount);
	printf("Kernel execution timeout:      %s\n", (devProp.kernelExecTimeoutEnabled ? "Yes" : "No"));
	return;
}

__global__ void kernelCalculateNewGenerationWithSharedMemory(float* current, float* next, int ny, int nx) {
	int ix = threadIdx.x + blockIdx.x * blockDim.x;
	int iy = threadIdx.y + blockIdx.y * blockDim.y;

	const float cx = 0.1;
	const float cy = 0.1;

	int me = ix + iy * nx,
		east = ix + 1 + iy * nx,
		west = ix - 1 + iy * nx,
		north = ix + (iy - 1) * nx,
		south = ix + (iy + 1) * nx;

	// INIT SHARED MEMORY
	__shared__ float dev_sharedMem[BLOCK_SIZE_Y][BLOCK_SIZE_X];
	dev_sharedMem[threadIdx.y][threadIdx.x] = current[me];

	__syncthreads();
	/* The point to update doesn't need an element that's "included" in this block */
	if ((threadIdx.x > 0) && (threadIdx.x < (BLOCK_SIZE_X - 1)) &&
		(threadIdx.y > 0) && (threadIdx.y < (BLOCK_SIZE_Y - 1))
		) {
		next[me] = cx * (dev_sharedMem[threadIdx.y][threadIdx.x-1] + dev_sharedMem[threadIdx.y][threadIdx.x+1] - 2.0f * dev_sharedMem[threadIdx.y][threadIdx.x]) +
			cy * (dev_sharedMem[threadIdx.y - 1][threadIdx.x] + dev_sharedMem[threadIdx.y + 1][threadIdx.x] - 2.0f * dev_sharedMem[threadIdx.y][threadIdx.x]) +
			dev_sharedMem[threadIdx.y][threadIdx.x];
	}
	else if (ix > 0 && ix < X_SIZE - 1 && iy > 0 && iy < Y_SIZE - 1) {
		next[me] =
			cx * (current[east] + current[west] - 2.0f * current[me]) +
			cy * (current[south] + current[north] - 2.0f * current[me]) +
			current[me];
	}
}

__global__ void kernelCalculateNewGeneration(float* current, float* next, int ny, int nx) {
	int ix = threadIdx.x + blockIdx.x * blockDim.x;
	int iy = threadIdx.y + blockIdx.y * blockDim.y;
	const float cx = 0.1;
	const float cy = 0.1;

	int me = ix + iy * nx,
		east = ix + 1 + iy * nx,
		west = ix - 1 + iy * nx,
		north = ix + (iy - 1) * nx,
		south = ix + (iy + 1) * nx;

	if (ix > 0 && ix < X_SIZE-1 && iy > 0 && iy < Y_SIZE-1) {
		next[me] =
			cx * (current[east] + current[west] - 2.0f * current[me]) +
			cy * (current[south] + current[north] - 2.0f * current[me]) +
			current[me];
	}
}

#define CEILDIV(a,b) (((a)+(b)-1)/(b))

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

int main() {
	float *dev_heatmap, *heatmap;
	float *dev_current_map, *dev_next_map;
	int iz;

	float duration = 0;
	hipEvent_t startEvent, endEvent;

	gpuErrchk(hipEventCreate(&startEvent));
	gpuErrchk(hipEventCreate(&endEvent));

	heatmap = (float*)malloc(ARRAY_SIZE*sizeof(float));

	printf("Grid is %dx%d and block is %dx%d\n", CEILDIV(X_SIZE, BLOCK_SIZE_X), CEILDIV(Y_SIZE, BLOCK_SIZE_Y), BLOCK_SIZE_X, BLOCK_SIZE_Y);

	// KERNEL CALL PARAMETRES INIT
	dim3 blockDim(BLOCK_SIZE_X, BLOCK_SIZE_Y);
	dim3 gridDim(CEILDIV(X_SIZE, BLOCK_SIZE_X), CEILDIV(Y_SIZE, BLOCK_SIZE_Y));

	// CPU ARRAY INITIALIZATION
	inidat(X_SIZE, Y_SIZE, heatmap);
	prtdat(X_SIZE, Y_SIZE, heatmap, input_file_name);

	// GPU INIT
	gpuErrchk(hipSetDevice(0));
	hipDeviceProp_t prop;
	gpuErrchk(hipGetDeviceProperties(&prop, 0));

	// Init timer to count the GPU processing time 
	// GPU processing time = Moving data from host to device + main loop (processing elements) + moving data from device to host
	hipEventRecord(startEvent);
	// GPU MEMORY INIT
	gpuErrchk(hipMalloc(&dev_heatmap, 2 * sizeof(float)*ARRAY_SIZE))
	gpuErrchk(hipMemcpy(dev_heatmap, heatmap, sizeof(float)*ARRAY_SIZE, hipMemcpyHostToDevice));
	memset(heatmap, '\0', sizeof(float)*ARRAY_SIZE);

	// PRE LOOP INITIALIZATIONS
	iz = 0;
	dev_current_map = dev_heatmap;
	dev_next_map = dev_heatmap + ARRAY_SIZE;

	// MAIN LOOP
	for (int t = 0 ; t < TIMESTEPS ; t++) {
		dev_current_map = dev_heatmap + ARRAY_SIZE * iz;
		dev_next_map = dev_heatmap + ARRAY_SIZE * (1 - iz);

		// KERNEL CALL
		//kernelCalculateNewGeneration<<<blockDim,gridDim>>>(dev_current_map,dev_next_map,Y_SIZE,X_SIZE);
		kernelCalculateNewGenerationWithSharedMemory<<<blockDim,gridDim >>>(dev_current_map, dev_next_map, Y_SIZE, X_SIZE);
		iz = 1 - iz;
	}

	gpuErrchk(hipMemcpy(heatmap, dev_next_map, sizeof(float)*ARRAY_SIZE, hipMemcpyDeviceToHost));
	gpuErrchk(hipEventRecord(endEvent));
	hipDeviceSynchronize();

	prtdat(X_SIZE, Y_SIZE, heatmap, output_file_name);
	gpuErrchk(hipEventElapsedTime(&duration, startEvent, endEvent));
	printf("GPU elapsed time: %f\n", duration);

	return 0;
}

void inidat(int nx, int ny, float *u) {
	int ix, iy;

	for (ix = 0; ix <= nx - 1; ix++)
		for (iy = 0; iy <= ny - 1; iy++)
			*(u + ix + nx * iy) = (float)(ix * (nx - ix - 1) * iy * (ny - iy - 1));
}

void prtdat(int nx, int ny, float *current, const char *fnam) {
	int ix, iy;
	FILE *fp;

	fp = fopen(fnam, "w");
	for (iy = 0; iy < Y_SIZE; iy++) {
		for (ix = 0; ix < nx; ix++) {
			fprintf(fp, "%6.1f", *(current + ix + nx*iy));
			if (ix != nx - 1)
				fprintf(fp, " ");
			else
				fprintf(fp, "\n");
		}
	}
	fclose(fp);
}


/*for (int t = 0; t < TIMESTEPS; t++) {
		hipError_t cudaStatus;
		dev_current_heatmap = dev_heatmap + iz * heatmap_size;
		dev_next_heatmap	= dev_heatmap + (1-iz) * heatmap_size;
		kernelCalculateNextIteration<<<dim3BlockSizes,dim3GridSizes>>>(dev_current_heatmap, dev_next_heatmap, Y_SIZE, X_SIZE, dev_someint);
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		}
		printf("Iteration %d\n", t);
		iz = 1 - iz;
	}*/
	//hipMemcpy(&someint, dev_someint, heatmap_size* sizeof(int), hipMemcpyDeviceToHost);